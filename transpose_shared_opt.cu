
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


// start with a small array to test
#define ROW 512
#define COL 512

__global__ void transpose(int *a, int *c, int nrow, int ncol){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    extern __shared__ int s[];
    if (i < ncol && j < nrow) {
        int grid32_x = threadIdx.x;
        int grid32_y = threadIdx.y;
        // printf("CUDA i %d j %d grid32_x %d grid32_y %d\n", i, j, grid32_x, grid32_y);
        s[grid32_x*(int)blockDim.y + grid32_y] = a[j*nrow + i];
        __syncthreads();
        int ci = threadIdx.x + blockIdx.y * blockDim.y;
        int cj = threadIdx.y + blockIdx.x * blockDim.x;
        c[cj*nrow + ci] = s[grid32_y*(int)blockDim.y + grid32_x];
    }
}

int main( void ){
    int a[ROW][COL];      // host copies of a, c
    int c[ROW][COL];
    int *dev_a;      // device copies of a, c (just pointers)
    int *dev_c;

    // get the size of the arrays I will need
    int size_2d = ROW * COL * sizeof(int);

    // Allocate the memory
    hipMalloc( (void**)&dev_a, size_2d);
    hipMalloc( (void**)&dev_c, size_2d);

    // Populate the 2D array on host with something small and known as a test
    for (int i=0; i < ROW; i++){
        for (int j=0; j < COL; j++){
            a[i][j] = i * COL + j;
            // printf("%d ", a[i][j]);
        }
        // printf("\n");
    }

    // Copy the memory
    hipMemcpy( dev_a, a, size_2d, hipMemcpyHostToDevice );
    // cudaMemcpy( dev_c, c, size_c, cudaMemcpyHostToDevice );

    // Run the kernal function
    dim3 tblocks(32, 32, 1);
    dim3 grid((COL/tblocks.x)+1, (ROW/tblocks.y)+1, 1);
    printf("Kernel launched with %d %d\n",grid.x, grid.y);
    transpose<<< grid, tblocks, tblocks.x*(tblocks.y+1)*sizeof(int) >>>(dev_a, dev_c, ROW, COL);
    auto transposeErr = hipGetLastError();
    if(transposeErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(transposeErr));
    
    auto asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
    // copy the output back to the host
    hipMemcpy( c, dev_c, size_2d, hipMemcpyDeviceToHost );

    // Print the output
    printf("\n");
    for (int i = 0; i < ROW; i++){
        for (int j=0; j < COL; j++){
            if (c[i][j] != a[j][i]) {
                printf("Error: (%d,%d) %d\n",i,j, c[i][j]);
            }
            // printf("%d ", c[i][j]);
        }
        // printf("\n");
    }
    // printf("right answer \n");
    // for (int i = 0; i < ROW; i++){
    //     for (int j=0; j < COL; j++){
    //         printf("%d ", a[j][i]);
    //     }
    //     printf("\n");
    // }

    // Releasae the memory
    hipFree( dev_a );
    hipFree( dev_c );
}

// Time (%)  Total Time (ns)  Instances  Avg (ns)  Med (ns)  Min (ns)  Max (ns)  StdDev (ns)     GridXYZ         BlockXYZ                   Name               
//  --------  ---------------  ---------  --------  --------  --------  --------  -----------  --------------  --------------  ---------------------------------
//     100.0            6’816          1   6’816.0   6’816.0     6’816     6’816          0.0    17   17    1    32   32    1  transpose(int *, int *, int, int)